#include "cuda_core/memory.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

// This file provides simple C-style wrappers around common CUDA memory operations.
// This helps to decouple the main application logic from direct CUDA API calls
// and provides a single place for error checking.

/**
 * @brief A helper macro to check for CUDA errors and print them.
 * This is a common pattern in CUDA programming. It makes error handling
 * much more concise.
 */
#define CUDA_CHECK(call) \
do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA Error in %s at line %d: %s\n", __FILE__, __LINE__, hipGetErrorString(err)); \
        return err; \
    } \
} while(0)

int cuda_malloc_device(void** devPtr, size_t size) {
    CUDA_CHECK(hipMalloc(devPtr, size));
    return hipSuccess;
}

int cuda_free_device(void* devPtr) {
    // hipFree can accept a NULL pointer, so we don't need to check.
    CUDA_CHECK(hipFree(devPtr));
    return hipSuccess;
}

int cuda_memcpy_host_to_device(void* dst, const void* src, size_t count) {
    CUDA_CHECK(hipMemcpy(dst, src, count, hipMemcpyHostToDevice));
    return hipSuccess;
}

int cuda_memcpy_device_to_host(void* dst, const void* src, size_t count) {
    CUDA_CHECK(hipMemcpy(dst, src, count, hipMemcpyDeviceToHost));
    return hipSuccess;
}
